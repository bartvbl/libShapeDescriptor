#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cassert>

#ifdef DESCRIPTOR_CUDA_KERNELS_ENABLED
#include <nvidia/hip/hip_runtime_api.h>
#endif

#include <shapeDescriptor/shapeDescriptor.h>
#ifdef DESCRIPTOR_CUDA_KERNELS_ENABLED
__inline__ __device__ float warpAllReduceSum(float val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
    return val;
}

__inline__ __device__ float computeDescriptorAverage(ShapeDescriptor::FPFHDescriptor &descriptor) {
    float threadSum = 0;
    for(unsigned int i = threadIdx.x; i < 3 * FPFH_BINS_PER_FEATURE; i += blockDim.x) {
        threadSum += descriptor.contents[i];
    }
    float totalSum = warpAllReduceSum(threadSum);
    return totalSum / float(3 * FPFH_BINS_PER_FEATURE);
}

__device__ float computeFPFHDescriptorSimilarity(
        ShapeDescriptor::FPFHDescriptor &needleDescriptor,
        float needleDescriptorAverage,
        ShapeDescriptor::FPFHDescriptor &haystackDescriptor) {

    float haystackDescriptorAverage = computeDescriptorAverage(haystackDescriptor);

    float threadSquaredSumX = 0;
    float threadSquaredSumY = 0;
    float threadMultiplicativeSum = 0;

    for(int i = threadIdx.x; i < 3 * FPFH_BINS_PER_FEATURE; i += warpSize) {
        float needleDescriptorValue = needleDescriptor.contents[i];
        float haystackDescriptorValue = haystackDescriptor.contents[i];

        float deltaX = float(needleDescriptorValue) - needleDescriptorAverage;
        float deltaY = float(haystackDescriptorValue) - haystackDescriptorAverage;

        threadSquaredSumX += deltaX * deltaX;
        threadSquaredSumY += deltaY * deltaY;
        threadMultiplicativeSum += deltaX * deltaY;
    }

    float squaredSumX = float(sqrt(warpAllReduceSum(threadSquaredSumX)));
    float squaredSumY = float(sqrt(warpAllReduceSum(threadSquaredSumY)));
    float multiplicativeSum = warpAllReduceSum(threadMultiplicativeSum);

    float correlation = multiplicativeSum / (squaredSumX * squaredSumY);

    return correlation;
}


__global__ void computeFPFHSearchResultIndices(
        ShapeDescriptor::FPFHDescriptor* needleDescriptors,
        ShapeDescriptor::FPFHDescriptor* haystackDescriptors,
        size_t haystackDescriptorCount,
        unsigned int* searchResults) {

#define needleDescriptorIndex blockIdx.x
    assert(blockDim.x == 32);

    __shared__ ShapeDescriptor::FPFHDescriptor referenceDescriptor;
    __shared__ ShapeDescriptor::FPFHDescriptor haystackDescriptor;

    for(unsigned int i = threadIdx.x; i < 3 * FPFH_BINS_PER_FEATURE; i += blockDim.x) {
        referenceDescriptor.contents[i] = needleDescriptors[needleDescriptorIndex].contents[i];
        haystackDescriptor.contents[i] = haystackDescriptors[needleDescriptorIndex].contents[i];
    }

    __syncthreads();

    float referenceDescriptorAverage = computeDescriptorAverage(referenceDescriptor);

    if(referenceDescriptorAverage == 0) {
        if(threadIdx.x == 0) {
            printf("WARNING: descriptor %i consists entirely of zeroes!\n", needleDescriptorIndex);
            // Effectively remove the descriptor from the list of search results
            atomicAdd(&searchResults[needleDescriptorIndex], 1 << 30);
        }
        return;
    }

    float referenceCorrelation = computeFPFHDescriptorSimilarity(
            referenceDescriptor,
            referenceDescriptorAverage,
            haystackDescriptor);

    // No image pair can have a better correlation than 1, so we can just stop the search right here
    if(referenceCorrelation == 1) {
        return;
    }

    unsigned int searchResultRank = 0;

    for(size_t haystackImageIndex = 0; haystackImageIndex < haystackDescriptorCount; haystackImageIndex++) {
        if(needleDescriptorIndex == haystackImageIndex) {
            continue;
        }

        for(unsigned int i = threadIdx.x; i < 3 * FPFH_BINS_PER_FEATURE; i += blockDim.x) {
            haystackDescriptor.contents[i] = haystackDescriptors[haystackImageIndex].contents[i];
        }

        float correlation = computeFPFHDescriptorSimilarity(
                referenceDescriptor,
                referenceDescriptorAverage,
                haystackDescriptor);

        // We've found a result that's better than the reference one. That means this search result would end up
        // above ours in the search result list. We therefore move our search result down by 1.
        if(correlation > referenceCorrelation) {
            searchResultRank++;
        }
    }

    if(threadIdx.x == 0) {
        atomicAdd(&searchResults[needleDescriptorIndex], searchResultRank);
    }
}
#endif

ShapeDescriptor::cpu::array<unsigned int> ShapeDescriptor::computeFPFHSearchResultRanks(
        ShapeDescriptor::gpu::array<ShapeDescriptor::FPFHDescriptor> device_needleDescriptors,
        ShapeDescriptor::gpu::array<ShapeDescriptor::FPFHDescriptor> device_haystackDescriptors,
        ShapeDescriptor::FPFHSearchExecutionTimes* executionTimes) {
#ifdef DESCRIPTOR_CUDA_KERNELS_ENABLED
    auto executionStart = std::chrono::steady_clock::now();

    size_t searchResultBufferSize = device_needleDescriptors.length * sizeof(unsigned int);
    unsigned int* device_searchResults;
    checkCudaErrors(hipMalloc(&device_searchResults, searchResultBufferSize));
    checkCudaErrors(hipMemset(device_searchResults, 0, searchResultBufferSize));

    auto searchStart = std::chrono::steady_clock::now();


    computeFPFHSearchResultIndices<<<device_needleDescriptors.length, 32>>>(
         device_needleDescriptors.content,
         device_haystackDescriptors.content,
         device_haystackDescriptors.length,
         device_searchResults);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    std::chrono::milliseconds searchDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - searchStart);

    ShapeDescriptor::cpu::array<unsigned int> resultIndices;
    resultIndices.content = new unsigned int[device_needleDescriptors.length];
    resultIndices.length = device_needleDescriptors.length;

    checkCudaErrors(hipMemcpy(resultIndices.content, device_searchResults, searchResultBufferSize, hipMemcpyDeviceToHost));

    // Cleanup
    checkCudaErrors(hipFree(device_searchResults));

    std::chrono::milliseconds executionDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - executionStart);

    if(executionTimes != nullptr) {
        executionTimes->searchExecutionTimeSeconds = double(searchDuration.count()) / 1000.0;
        executionTimes->totalExecutionTimeSeconds = double(executionDuration.count()) / 1000.0;
    }

    return resultIndices;
#else
    throw std::runtime_error(ShapeDescriptor::cudaMissingErrorMessage);
#endif
}







#ifdef DESCRIPTOR_CUDA_KERNELS_ENABLED
__global__ void computeElementWiseFPFHEuclideanDistances(
        ShapeDescriptor::FPFHDescriptor* descriptors,
        ShapeDescriptor::FPFHDescriptor* correspondingDescriptors,
        float* distances) {
    const size_t descriptorIndex = blockIdx.x;

    static_assert(spinImageWidthPixels % 32 == 0, "This kernel assumes the image is a multiple of the warp size wide");

    size_t needleImageIndex = blockIdx.x;

    float threadSquaredSum = 0;

    for(unsigned int i = threadIdx.x; i < 3 * FPFH_BINS_PER_FEATURE; i += blockDim.x) {
        float descriptorPixelValue = descriptors[needleImageIndex].contents[i];
        float correspondingPixelValue = correspondingDescriptors[needleImageIndex].contents[i];
        float pixelDelta = descriptorPixelValue - correspondingPixelValue;
        threadSquaredSum += pixelDelta * pixelDelta;
    }

    float totalSquaredSum = warpAllReduceSum(threadSquaredSum);

    if(threadIdx.x == 0) {
        distances[descriptorIndex] = sqrt(totalSquaredSum);
    }
}
#endif

ShapeDescriptor::cpu::array<float> ShapeDescriptor::computeFPFHElementWiseEuclideanDistances(
        ShapeDescriptor::gpu::array<ShapeDescriptor::FPFHDescriptor> device_descriptors,
        ShapeDescriptor::gpu::array<ShapeDescriptor::FPFHDescriptor> device_correspondingDescriptors) {
    ShapeDescriptor::gpu::array<float> device_distances(device_descriptors.length);

    computeElementWiseFPFHEuclideanDistances<<<device_descriptors.length, 32>>>(
            device_descriptors.content,
            device_correspondingDescriptors.content,
            device_distances.content);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    ShapeDescriptor::cpu::array<float> distances = device_distances.copyToCPU();

    ShapeDescriptor::free(device_distances);

    return distances;
}