#include "hip/hip_runtime.h"
#include "meshSampler.cuh"

#include <shapeDescriptor/common/types/SampleBounds.h>
#include <shapeDescriptor/gpu/types/CudaLaunchDimensions.h>

#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <shapeDescriptor/gpu/types/array.h>

#include "nvidia/hip/hip_vector_types.h"
#include "nvidia/hip/hip_runtime_api.h"

#define SAMPLE_COEFFICIENT_THREAD_COUNT 4096

__device__ __inline__ ShapeDescriptor::SampleBounds calculateSampleBounds(const ShapeDescriptor::gpu::array<float> &areaArray, int triangleIndex, int sampleCount) {
    ShapeDescriptor::SampleBounds sampleBounds;
    float maxArea = areaArray.content[areaArray.length - 1];
    float areaStepSize = maxArea / (float)sampleCount;

    if (triangleIndex == 0)
    {
        sampleBounds.areaStart = 0;
        sampleBounds.areaEnd = areaArray.content[0];
    }
    else
    {
        sampleBounds.areaStart = areaArray.content[triangleIndex - 1];
        sampleBounds.areaEnd = areaArray.content[triangleIndex];
    }

    size_t firstIndexInRange = (size_t) (sampleBounds.areaStart / areaStepSize) + 1;
    size_t lastIndexInRange = (size_t) (sampleBounds.areaEnd / areaStepSize);

    sampleBounds.sampleCount = lastIndexInRange - firstIndexInRange + 1; // Offset is needed to ensure bounds are correct
    sampleBounds.sampleStartIndex = firstIndexInRange - 1;

    return sampleBounds;
}

__device__ __inline__ void lookupTriangleVertices(ShapeDescriptor::gpu::Mesh mesh, int triangleIndex, float3 (&triangleVertices)[3]) {
    assert(triangleIndex >= 0);
    assert((3 * triangleIndex) + 2 < mesh.vertexCount);

    unsigned int triangleBaseIndex = 3 * triangleIndex;

    triangleVertices[0].x = mesh.vertices_x[triangleBaseIndex];
    triangleVertices[0].y = mesh.vertices_y[triangleBaseIndex];
    triangleVertices[0].z = mesh.vertices_z[triangleBaseIndex];

    triangleVertices[1].x = mesh.vertices_x[triangleBaseIndex + 1];
    triangleVertices[1].y = mesh.vertices_y[triangleBaseIndex + 1];
    triangleVertices[1].z = mesh.vertices_z[triangleBaseIndex + 1];

    triangleVertices[2].x = mesh.vertices_x[triangleBaseIndex + 2];
    triangleVertices[2].y = mesh.vertices_y[triangleBaseIndex + 2];
    triangleVertices[2].z = mesh.vertices_z[triangleBaseIndex + 2];
}

__device__ __inline__ void lookupTriangleNormals(ShapeDescriptor::gpu::Mesh mesh, int triangleIndex, float3 (&triangleNormals)[3]) {
    assert(triangleIndex >= 0);
    assert((3 * triangleIndex) + 2 < mesh.vertexCount);

    unsigned int triangleBaseIndex = 3 * triangleIndex;

    triangleNormals[0].x = mesh.normals_x[triangleBaseIndex];
    triangleNormals[0].y = mesh.normals_y[triangleBaseIndex];
    triangleNormals[0].z = mesh.normals_z[triangleBaseIndex];

    triangleNormals[1].x = mesh.normals_x[triangleBaseIndex + 1];
    triangleNormals[1].y = mesh.normals_y[triangleBaseIndex + 1];
    triangleNormals[1].z = mesh.normals_z[triangleBaseIndex + 1];

    triangleNormals[2].x = mesh.normals_x[triangleBaseIndex + 2];
    triangleNormals[2].y = mesh.normals_y[triangleBaseIndex + 2];
    triangleNormals[2].z = mesh.normals_z[triangleBaseIndex + 2];
}


// One thread = One triangle
__global__ void calculateAreas(ShapeDescriptor::gpu::array<float> areaArray, ShapeDescriptor::gpu::Mesh mesh)
{
    int triangleIndex = blockDim.x * blockIdx.x + threadIdx.x;
    if (triangleIndex >= areaArray.length)
    {
        return;
    }
    float3 vertices[3];
    lookupTriangleVertices(mesh, triangleIndex, vertices);
    float3 v1 = vertices[1] - vertices[0];
    float3 v2 = vertices[2] - vertices[0];
    float area = length(cross(v1, v2)) / 2.0;
    areaArray.content[triangleIndex] = area;
}

__global__ void calculateCumulativeAreas(ShapeDescriptor::gpu::array<float> areaArray, ShapeDescriptor::gpu::array<float> device_cumulativeAreaArray) {
    int triangleIndex = blockDim.x * blockIdx.x + threadIdx.x;
    if (triangleIndex >= areaArray.length)
    {
        return;
    }

    float totalArea = 0;

    for(int i = 0; i <= triangleIndex; i++) {
        // Super inaccurate. Don't try this at home.
        totalArea += areaArray.content[i];
    }

    device_cumulativeAreaArray.content[triangleIndex] = totalArea;
}

__global__ void generateRandomSampleCoefficients(ShapeDescriptor::gpu::array<float2> coefficients, hiprandState *randomState, int sampleCount, size_t randomSeed) {
    int rawThreadIndex = threadIdx.x+blockDim.x*blockIdx.x;

    assert(rawThreadIndex < SAMPLE_COEFFICIENT_THREAD_COUNT);

    if(randomSeed == 0) {
        randomSeed = clock64();
    }

    // The addition of the thread index is overkill, but whatever. Randomness!
    size_t skipFactor = rawThreadIndex + (gridDim.x * blockDim.x);

    hiprand_init(randomSeed, skipFactor, 0, &randomState[rawThreadIndex]);

    for(int i = rawThreadIndex; i < sampleCount; i += blockDim.x * gridDim.x) {
        float v1 = hiprand_uniform(&(randomState[rawThreadIndex]));
        float v2 = hiprand_uniform(&(randomState[rawThreadIndex]));

        coefficients.content[i].x = v1;
        coefficients.content[i].y = v2;
    }
}

// One thread = One triangle
__global__ void sampleMesh(
        ShapeDescriptor::gpu::Mesh mesh,
        ShapeDescriptor::gpu::array<float> areaArray,
        ShapeDescriptor::gpu::PointCloud pointCloud,
        ShapeDescriptor::gpu::array<float2> coefficients,
        int sampleCount) {
    int triangleIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(triangleIndex >= mesh.vertexCount / 3)
    {
        return;
    }

    float3 triangleVertices[3];
    lookupTriangleVertices(mesh, triangleIndex, triangleVertices);

    float3 triangleNormals[3];
    lookupTriangleNormals(mesh, triangleIndex, triangleNormals);

    ShapeDescriptor::SampleBounds bounds = calculateSampleBounds(areaArray, triangleIndex, sampleCount);

    for(int sample = 0; sample < bounds.sampleCount; sample++) {
        size_t sampleIndex = bounds.sampleStartIndex + sample;

        if(sampleIndex >= sampleCount) {
            continue;
        }

        float v1 = coefficients.content[sampleIndex].x;
        float v2 = coefficients.content[sampleIndex].y;

        float3 samplePoint =
                (1 - sqrt(v1)) * triangleVertices[0] +
                (sqrt(v1) * (1 - v2)) * triangleVertices[1] +
                (sqrt(v1) * v2) * triangleVertices[2];

        float3 sampleNormal =
                (1 - sqrt(v1)) * triangleNormals[0] +
                (sqrt(v1) * (1 - v2)) * triangleNormals[1] +
                (sqrt(v1) * v2) * triangleNormals[2];
        sampleNormal = normalize(sampleNormal);

        assert(sampleIndex < sampleCount);
        pointCloud.vertices.set(sampleIndex, samplePoint);
        pointCloud.normals.set(sampleIndex, sampleNormal);
    }
}

ShapeDescriptor::gpu::PointCloud ShapeDescriptor::utilities::sampleMesh(gpu::Mesh device_mesh, size_t sampleCount, size_t randomSamplingSeed, ShapeDescriptor::internal::MeshSamplingBuffers* internalSampleBuffers) {
    size_t vertexCount = device_mesh.vertexCount;
    size_t triangleCount = vertexCount / 3;

    size_t areaArrayLength = triangleCount;
    size_t areaArraySize = areaArrayLength * sizeof(float);
    hiprandState* device_randomState;
    ShapeDescriptor::gpu::array<float2> device_coefficients;

    ShapeDescriptor::gpu::array<float> device_areaArray;
    ShapeDescriptor::gpu::array<float> device_cumulativeAreaArray;

    gpu::PointCloud device_pointCloud(sampleCount);

    checkCudaErrors(hipMalloc(&device_areaArray.content, areaArraySize));
    checkCudaErrors(hipMalloc(&device_cumulativeAreaArray.content, areaArraySize));
    checkCudaErrors(hipMalloc(&device_randomState, sizeof(hiprandState) * (size_t)SAMPLE_COEFFICIENT_THREAD_COUNT));
    checkCudaErrors(hipMalloc(&device_coefficients.content, sizeof(float2) * sampleCount));

    device_areaArray.length = (size_t) areaArrayLength;
    device_cumulativeAreaArray.length = (size_t) areaArrayLength;

    gpu::CudaLaunchDimensions areaSettings = calculateCudaLaunchDimensions(device_areaArray.length);
    gpu::CudaLaunchDimensions cumulativeAreaSettings = calculateCudaLaunchDimensions(device_areaArray.length);

    calculateAreas <<<areaSettings.blocksPerGrid, areaSettings.threadsPerBlock >>> (device_areaArray, device_mesh);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    calculateCumulativeAreas<<<cumulativeAreaSettings.blocksPerGrid, cumulativeAreaSettings.threadsPerBlock>>>(device_areaArray, device_cumulativeAreaArray);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    generateRandomSampleCoefficients<<<SAMPLE_COEFFICIENT_THREAD_COUNT / 32, 32>>>(device_coefficients, device_randomState, sampleCount, randomSamplingSeed);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    sampleMesh <<<areaSettings.blocksPerGrid, areaSettings.threadsPerBlock>>>(device_mesh, device_cumulativeAreaArray, device_pointCloud, device_coefficients, sampleCount);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    if(internalSampleBuffers != nullptr) {
        internalSampleBuffers->cumulativeAreaArray = device_cumulativeAreaArray;
    } else {
        hipFree(device_cumulativeAreaArray.content);
    }

    hipFree(device_areaArray.content);
    hipFree(device_randomState);
    hipFree(device_coefficients.content);

    return device_pointCloud;
}